#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.cuh"
#include "TurboFNO.h"
#include "fused_cgemm_ifft_7.cuh"
#include "fused_cgemm_ifft_8.cuh"
#include "fused_cgemm_ifft_9.cuh"
#include "fused_cgemm_ifft_10.cuh"
#include "fft_radix_2_logN_7_upload_0_stride_DY.cuh"
#include "fft_radix_2_logN_8_upload_0_stride_DY.cuh"
#include "fft_radix_2_logN_9_upload_0_stride_DY.cuh"
#include "fft_radix_2_logN_10_upload_0_stride_DY.cuh"
#include <hipfft/hipfftXt.h>


using DataT = float2;
int thread_bs[4] = {8, 16, 8, 16};
void (*fused_cgemm_ifft [4])(int, int, int, float2 *, float2 *, float2 *, float2 *, float2, float2) = 
{fused_cgemm_ifft_7, fused_cgemm_ifft_8, fused_cgemm_ifft_9, fused_cgemm_ifft_10};
void (*fft_stride_DY [4])(float2 *, float2 *, int, int) = {fft_7_stride_DY, fft_8_stride_DY, fft_9_stride_DY, fft_10_stride_DY};



int main(int argc, char** argv){
      DataT *A, *dA, *B, *dB, *C, *C_ref, *dC, *dC_ref, 
            *FFT_input, *dFFT_input, *dFFT_output, 
            *iFFT_output, *diFFT_output, *iFFT_output_ref, *diFFT_output_ref;
    long long int bs, dimX, dimY, DY, M, N, K, FFT_len, FFT_bs, iFFT_bs, FFT_input_size, iFFT_output_size;
    bs = 128;
    dimX = 256;
    DY = 256;
    N =  128;
    K = 128;
    ntest = 5;

        // 解析命令行参数
        if (argc > 1) {
          if (argc != 6) {
              printf("Usage: %s <bs> <dimX> <DY> <N> <K>\n", argv[0]);
              printf("Example: %s 128 256 256 128 128\n", argv[0]);
              printf("Using default values: bs=%lld, dimX=%lld, DY=%lld, N=%lld, K=%lld\n", 
                     bs, dimX, DY, N, K);
          } else {
              bs = atoi(argv[1]);
              dimX = atoi(argv[2]);
              DY = atoi(argv[3]);
              N = atoi(argv[4]);
              K = atoi(argv[5]);
              
          }
      }

    M = bs * dimX * THREADBLOCK_M;
    dimY = 64;
    FFT_len = DY;
    FFT_bs = bs * dimX * K;
    iFFT_bs = bs * dimX * N;
    FFT_input_size = bs * dimX * DY * K;
    iFFT_output_size = bs * dimX * DY * N;

      long long int A_size = M * K;
      long long int B_size = N * K;
      long long int C_size = M * N;
      FFT_input = (DataT*)malloc(sizeof(DataT) * (FFT_input_size + ntest));
      iFFT_output = (DataT*)malloc(sizeof(DataT) * (iFFT_output_size + ntest));
      iFFT_output_ref = (DataT*)malloc(sizeof(DataT) * (iFFT_output_size + ntest));
      B = (DataT*)malloc(sizeof(DataT) * (B_size + ntest));
      C = (DataT*)malloc(sizeof(DataT) * (C_size + ntest));
      C_ref = (DataT*)malloc(sizeof(DataT) * (C_size + ntest));


      CUDA_RT_CALL(hipMalloc((void**)&dFFT_input, sizeof(DataT) * (FFT_input_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&dFFT_output, sizeof(DataT) * (FFT_input_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&diFFT_output, sizeof(DataT) * (iFFT_output_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&diFFT_output_ref, sizeof(DataT) * (iFFT_output_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&dA, sizeof(DataT) * (A_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&dB, sizeof(DataT) * (B_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&dC, sizeof(DataT) * (C_size + ntest)));
      CUDA_RT_CALL(hipMalloc((void**)&dC_ref, sizeof(DataT) * (C_size + ntest)));

      generate_random_vector((float*)FFT_input, FFT_input_size * 2);
      generate_random_vector((float*)B, B_size * 2);
      fill_vector((float*)C, 0, C_size * 2);
      fill_vector((float*)iFFT_output, 0, iFFT_output_size * 2);

      CUDA_RT_CALL(hipMemcpy(dFFT_input, FFT_input, sizeof(DataT) * FFT_input_size, hipMemcpyHostToDevice));
      CUDA_RT_CALL(hipMemcpy(dFFT_output, FFT_input, sizeof(DataT) * FFT_input_size, hipMemcpyHostToDevice));

      CUDA_RT_CALL(hipMemcpy(diFFT_output, iFFT_output, sizeof(DataT) * iFFT_output_size, hipMemcpyHostToDevice));
      CUDA_RT_CALL(hipMemcpy(diFFT_output_ref, iFFT_output, sizeof(DataT) * iFFT_output_size, hipMemcpyHostToDevice));
      
      CUDA_RT_CALL(hipMemcpy(dB, B, sizeof(DataT) * B_size, hipMemcpyHostToDevice));
      CUDA_RT_CALL(hipMemcpy(dC, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice));
      CUDA_RT_CALL(hipMemcpy(dC_ref, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice));

      DataT alpha = {1.0, -1.0} , beta = {-1.0, 1.0}; 
      // DataT alpha = {1.0, 0} , beta = {1.0, 0}; 

            
      // std::ifstream infile(DEFAULT_CONFIG_PATH );
      // std::string line;
  
      // std::unordered_map<std::string, std::vector<int>> config;
  
      // while (std::getline(infile, line)) {
      //     if (line.empty() || line[0] == '#') continue;  // 跳过注释或空行
      //     std::istringstream iss(line);
      //     std::string key;
      //     iss >> key;
      //     config[key] = parse_line(line);
      // }
  
      // // 提取参数
      // auto& bs_list   = config["bs_list"];
      // auto& dimX_list = config["dimX_list"];
      // auto& DY_list   = config["DY_list"];
      // auto& N_list    = config["N_list"];
      // auto& K_list    = config["K_list"];
      
      // for (int bs : bs_list) {
      //   for (int dimX : dimX_list) {
      //       for (int DY : DY_list) {
      //           for (int N : N_list) {
      //               for (int K : K_list) {
      M = bs * dimX * THREADBLOCK_M;
      dimY = 64;
      FFT_len = DY;
      FFT_bs = bs * dimX * K;
      iFFT_bs = bs * dimX * N;
      FFT_input_size = bs * dimX * DY * K;
      iFFT_output_size = bs * dimX * DY * N;

      A_size = M * K;
      B_size = N * K;
      C_size = M * N;
      
      CUDA_RT_CALL(hipMemcpy(dFFT_input, FFT_input, sizeof(DataT) * FFT_input_size, hipMemcpyHostToDevice));
                
      dim3 gridDim((M + THREADBLOCK_M - 1) / THREADBLOCK_M, (N + THREADBLOCK_N - 1) / THREADBLOCK_N, 1);
      dim3 blockDim((THREADBLOCK_M * THREADBLOCK_N / (THREAD_M * THREAD_N)), 1, 1); 
      int shmem_size = sizeof(DataT) * (THREADBLOCK_M + THREADBLOCK_N) * THREADBLOCK_K * 2;
      shmem_size = shmem_size > sizeof(DataT) * (THREADBLOCK_M + THREADBLOCK_N + DY) * THREADBLOCK_K ? shmem_size : sizeof(DataT)  * (THREADBLOCK_M + THREADBLOCK_N + DY) * THREADBLOCK_K;
      
      
      hipDeviceSynchronize();
      int logFFT_len = int(log2f(DY)) - 7;
      dim3 gridDim_fft_dimY((dimX * K * bs + threadblock_bs - 1) / threadblock_bs, 1, 1);
      dim3 gridDim_ifft_dimY((dimX * N * bs + threadblock_bs - 1) / threadblock_bs, 1, 1);
      gridDim_fft_dimY.x = gridDim_fft_dimY.x > 65536 ? 65536 : gridDim_fft_dimY.x;
      gridDim_ifft_dimY.x = gridDim_ifft_dimY.x > 65536 ? 65536 : gridDim_ifft_dimY.x;
      dim3 blockDim_fft_dimY(DY / thread_bs[logFFT_len] * threadblock_bs, 1, 1); 
      int shmem_size_fft_dimY = sizeof(DataT) * DY * threadblock_bs ;  
      
      fft_stride_DY[logFFT_len]<<<gridDim_fft_dimY, blockDim_fft_dimY, shmem_size_fft_dimY>>>(dFFT_input,  dA, threadblock_bs, dimX * K * bs);
      CHECK_CUDA_KERNEL();
      hipDeviceSynchronize();
      CHECK_CUDA_KERNEL();
      
      fused_cgemm_ifft[logFFT_len]<<<gridDim, blockDim, shmem_size>>>(M, N, K, dA, dB, dC, diFFT_output, alpha, beta);
      CHECK_CUDA_KERNEL();
      hipDeviceSynchronize();
      CHECK_CUDA_KERNEL();


      {
        hipEvent_t fft_begin, fft_end;
        float elapsed_time;
        hipEventCreate(&fft_begin);
        hipEventCreate(&fft_end);
      hipEventRecord(fft_begin);
      for (int i = 0; i < ntest; ++i){
        fft_stride_DY[logFFT_len]<<<gridDim_fft_dimY, blockDim_fft_dimY, shmem_size_fft_dimY>>>(dFFT_input, dA, threadblock_bs, dimX * K * bs);
        hipDeviceSynchronize();
        fused_cgemm_ifft[logFFT_len]<<<gridDim, blockDim, shmem_size>>>(M, N, K, dA, dB, dC, diFFT_output, alpha, beta);
        hipDeviceSynchronize();
      }
      hipEventRecord(fft_end);
      hipEventSynchronize(fft_begin);
      hipEventSynchronize(fft_end);
      hipEventElapsedTime(&elapsed_time, fft_begin, fft_end);

      elapsed_time = elapsed_time / ntest;
      printf("1D_C, bs=%-4d, dimX=%-4d, DY=%-4d, N=%-4d, K=%-4d, TIME=%8.3fms\n",
        bs, dimX, DY, N, K, elapsed_time);
    }
  // }}}}}
    return 0;
}