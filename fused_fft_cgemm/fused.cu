#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.cuh"
#include "fused_fft_cgemm.cuh"
#include <hipfft/hipfftXt.h>

using DataT = float2;
int thread_bs[4] = {8, 16, 8, 16};

// Kernel to truncate the top 'reduced_M' rows of an (M x K) column-major matrix of float2
// by copying them in chunks of float4 (2 x float2). 
__global__ void direct_copy_colmajor_float4(const float2 *input, 
  float2       *output,
  int M, int K, int reduced_M)
{
// Reinterpret float2* as float4* (each float4 = 2 float2)
const float4 *inputF4  = reinterpret_cast<const float4 *>(input);
float4       *outputF4 = reinterpret_cast<float4       *>(output);

// Number of float4 elements per column in the *original* M rows
int inputFloat4PerCol  = M / 2; 
// Number of float4 elements per column for the truncated (reduced_M) rows
int outputFloat4PerCol = reduced_M / 2;

// 2D block & grid:
//   - 'col' indexes the columns (up to K)
//   - 'row4' indexes the float4 chunks along the row dimension (up to reduced_M/2)
int col  = blockIdx.y * blockDim.y + threadIdx.y;  // column index [0..K-1]
int row4 = blockIdx.x * blockDim.x + threadIdx.x;  // float4 index [0..(reduced_M/2)-1]

// Bounds check
if (col < K && row4 < outputFloat4PerCol) {
// In column-major, the float4 index for the input is:
//      inIndex  = col * (M/2) + row4
// For the output (which has reduced_M rows):
//      outIndex = col * (reduced_M/2) + row4

int inIndex  = col * inputFloat4PerCol  + row4;
int outIndex = col * outputFloat4PerCol + row4;

// Copy one float4 (which is 2 float2)
outputF4[outIndex] = inputF4[inIndex];
}
}


int main(int argc, char** argv){
    if(argc < 7){
      printf("Usage: %s bs dimX dimY N K ntest\n", argv[0]);
      return 1;
    }
      DataT *A, *dA, *B, *dB, *C, *C_ref, *dC, *dC_ref, *FFT_input, *dFFT_input, *FFT_output, *dFFT_output;
    int bs, dimX, dimY, M, N, K, FFT_len, FFT_bs, FFT_input_size;
      bs = atoi(argv[1]);
      dimX = atoi(argv[2]);
      dimY = atoi(argv[3]);
      N = atoi(argv[4]);
      K = atoi(argv[5]);
      M = bs * dimX * THREADBLOCK_M;
      FFT_len = dimY;
      FFT_bs = bs * dimX * K;
      FFT_input_size = bs * dimX * dimY * K;
      printf("bs=%d dimX=%d dimY=%d M=%d, N=%d K=%d\n", bs, dimX, dimY, M, N, K);
      printf("FFT_len=%d FFT_bs=%d\n", FFT_len, FFT_bs);
      int num_tests = atoi(argv[6]);

      long long int A_size = M * K;
      long long int B_size = N * K;
      long long int C_size = M * N;
      FFT_input = (DataT*)malloc(sizeof(DataT) * FFT_input_size);
      B = (DataT*)malloc(sizeof(DataT) * B_size);
      C = (DataT*)malloc(sizeof(DataT) * C_size);
      C_ref = (DataT*)malloc(sizeof(DataT) * C_size);

      hipMalloc((void**)&dFFT_input, sizeof(DataT) * FFT_input_size);
      hipMalloc((void**)&dFFT_output, sizeof(DataT) * FFT_input_size);
      hipMalloc((void**)&dA, sizeof(DataT) * A_size);
      hipMalloc((void**)&dB, sizeof(DataT) * B_size);
      hipMalloc((void**)&dC, sizeof(DataT) * C_size);
      hipMalloc((void**)&dC_ref, sizeof(DataT) * C_size);

      generate_random_vector((float*)FFT_input, FFT_input_size * 2);
      generate_random_vector((float*)B, B_size * 2);
      fill_vector((float*)C, 0, C_size * 2);

      hipMemcpy(dFFT_input, FFT_input, sizeof(DataT) * FFT_input_size, hipMemcpyHostToDevice);
      hipMemcpy(dB, B, sizeof(DataT) * B_size, hipMemcpyHostToDevice);
      hipMemcpy(dC, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice);
      hipMemcpy(dC_ref, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice);

      DataT alpha = {1.0, -1.0} , beta = {-1.0, 1.0}; 
      // DataT alpha = {1.0, 0} , beta = {1.0, 0}; 

      
      dim3 gridDim((M + THREADBLOCK_M - 1) / THREADBLOCK_M, (N + THREADBLOCK_N - 1) / THREADBLOCK_N, 1);
      dim3 blockDim((THREADBLOCK_M * THREADBLOCK_N / (THREAD_M * THREAD_N)), 1, 1); 
      int shmem_size = sizeof(DataT) * (THREADBLOCK_M + THREADBLOCK_N + dimY) * THREADBLOCK_K ;  
      printf("blockDim .x=%d .y=%d .z=%d\n", blockDim.x, blockDim.y, blockDim.z);
      printf("gridDim .x=%d .y=%d .z=%d\n", gridDim.x, gridDim.y, gridDim.z);
      printf("shmem size = %d byte\n", shmem_size);
      dim3 blockDim_copy(256, 1, 1);

      // The grid:
      //  - x-dim covers the range of row4 in [0, outputFloat4PerCol)
      //  - y-dim covers the range of columns in [0, K)
      dim3 gridDim_copy((THREADBLOCK_M / 2 + blockDim_copy.x - 1) / blockDim_copy.x,
                   (FFT_bs + blockDim_copy.y - 1) / blockDim_copy.y);

      

      hipblasHandle_t handle;   
      hipblasCreate(&handle);

      hipfftHandle plan;
      hipfftCreate(&plan);
  
      hipfftPlan1d(&plan, FFT_len, HIPFFT_C2C, FFT_bs);

      printf("start cuFFT!\n");
      hipfftExecC2C(plan, reinterpret_cast<hipfftComplex*>(dFFT_input), 
                          reinterpret_cast<hipfftComplex*>(dFFT_output), 
                          HIPFFT_FORWARD);
      hipDeviceSynchronize();
      printf("start copy!\n");
      fflush(stdout);
      direct_copy_colmajor_float4<<<gridDim_copy, blockDim_copy>>>(dFFT_output, dA, FFT_len, FFT_bs, THREADBLOCK_M);
      hipError_t err = hipGetLastError();  // 获取最近的错误
      if (err != hipSuccess) {
          printf("CUDA Kernel Launch Error: %s\n", hipGetErrorString(err));
      }
      hipDeviceSynchronize();
      printf("Finish copy!\n");
      err = hipGetLastError();  // 获取最近的错误
      if (err != hipSuccess) {
          printf("CUDA Kernel Launch Error: %s\n", hipGetErrorString(err));
      }
      hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, (hipFloatComplex*)&alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, (hipFloatComplex*)&beta, (hipFloatComplex*)dC_ref, M);     
      hipDeviceSynchronize();


      printf("Start Fused!\n");
      fused_fft_cgemm<<<gridDim, blockDim, shmem_size>>>(M, N, K, dFFT_input, dB, dC, alpha, beta);
      
      err = hipGetLastError();  // 获取最近的错误
      if (err != hipSuccess) {
          printf("CUDA Kernel Launch Error: %s\n", hipGetErrorString(err));
      }
      hipDeviceSynchronize();
      printf("Finish Fused!\n");
      err = hipGetLastError();  // 获取最近的错误
      if (err != hipSuccess) {
          printf("CUDA Kernel Launch Error: %s\n", hipGetErrorString(err));
      }
        
    

      hipMemcpy(C, dC, sizeof(DataT) * C_size, hipMemcpyDeviceToHost);
      hipMemcpy(C_ref, dC_ref, sizeof(DataT) * C_size, hipMemcpyDeviceToHost);

      verify_vector((float*)C_ref, (float*)C, M * N * 2, M);

    return 0;
}