#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>  
#include "utils/utils.cuh"
#include "kernels.cuh"       
#define PPP 1    
#include <hip/hip_runtime.h> 
#include <helper_functions.h>           
#include <hip/hip_runtime_api.h>  
#define BM 64
#define BN 64
#define BK 8

int main(int argc, char **argv)
{        
    if (argc < 2) {  
        printf("Please select a kernel (range 0 - 1, here 0 is for NVIDIA cuBLAS).\n");
         exit(-1);
    }
    srand(10);  
    int kernel_number = atoi(argv[1]);
    int num_tests = 10;
    const int NSPLIT = atoi(argv[4]);
    int start_size = atoi(argv[2]);   
    int end_size = atoi(argv[3]); 
    int gap_size = 256;
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
        printf("%8.2d|", max_size);
    }
    printf("\n");  
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
        printf("%8.2f|", min(8.1, double(max_size) * 31.25 / 1e3));
    } 

    printf("\n");
    int threads_x = atoi(argv[4]); 
    double2 alpha, beta; 
    alpha.x = 1.0, alpha.y = 1.0;
	beta.x = 1.0, beta.y = 1.0;  
    int max_size = end_size;
    double *A = NULL, *B = NULL, *C_ref = NULL, *C = NULL;
    double *dA = NULL,*dB = NULL, *dC_ref = NULL, *dC = NULL;
    int size = max_size * sizeof (int);
    int deviceId;
    hipGetDevice(&deviceId); 
    hipDeviceProp_t props = getDetails(deviceId);
    
    A = (double *)malloc(sizeof(double) * max_size * max_size * 2);
    B = (double *)malloc(sizeof(double) * max_size * max_size * 2);
    C = (double *)malloc(sizeof(double) * max_size * max_size * 2);
    C_ref = (double *)malloc(sizeof(double) * max_size * max_size * 2);
    
    generate_random_matrix_double(A, max_size);
    generate_random_matrix_double(A + max_size * max_size, max_size);
     
    generate_random_matrix_double(B, max_size);
    generate_random_matrix_double(B + max_size * max_size, max_size);
    
    generate_random_matrix_double(C, max_size);
    generate_random_matrix_double(C + max_size * max_size, max_size);
    
    // for(int i = 0; i < max_size; ++i){
    //     for(int j = 0; j < max_size; ++j){  
    //         C[j + i * max_size] = (double)(j + i * max_size);
    //         C[j + i * max_size + max_size * max_size] = (double)(j + i * max_size);
    //         A[j * 2 + i * max_size * 2] = (double)j;
    //         A[j * 2 + i * max_size * 2 + 1] = (double)i;
    //         B[j * 2 + i * max_size * 2] = (double)j;
    //         B[j * 2 + i * max_size * 2 + 1] = (double)i;
 
    //     }
    // }
    copy_matrix_double(C, C_ref, max_size); 
    copy_matrix_double(C + max_size * max_size, C_ref + max_size * max_size , max_size);
   
    CUDA_CALLER(hipMalloc((void**) &dA, sizeof(double) * max_size * max_size * 2));
    CUDA_CALLER(hipMalloc((void**) &dB, sizeof(double) * max_size * max_size * 2));
    CUDA_CALLER(hipMalloc((void**) &dC, sizeof(double) * max_size * max_size * 2));
    CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(double) * max_size * max_size * 2));
      
    CUDA_CALLER(hipMemcpy(dA, A, sizeof(double) * max_size * max_size * 2, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB, B, sizeof(double) * max_size * max_size * 2, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC, C, sizeof(double) * max_size * max_size * 2, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC_ref, C_ref, sizeof(double) * max_size * max_size * 2, hipMemcpyHostToDevice));

    hipblasHandle_t handle;   
    hipblasCreate(&handle);      
      
    // if (!verify_matrix_double(C_ref, C, max_size) ||  
    //     !verify_matrix_double(C_ref + max_size * max_size, C + max_size * max_size, max_size)) {
    //     printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
    //     exit(-3);  
    // }    
    int M, N, K;        
    int shared_mem_size = ((BM + SKEW_KERNEL_2) * BK * 2 + (BN + SKEW_KERNEL_2) * BK * 2 ) * 2 * 8; 
    shared_mem_size = ((64 + 4) * 16 * 2) * 3 * 8;
    int shared_mem_size_18 = ((64 + 0) * 16 * 2) * 3 * 8;
    int shared_mem_size_64x128x8 = ((64 + 128) * 16) * 3 * 8;
    // shared_mem_size = 76800; 
    // hipDeviceSetSharedMemConfig( hipSharedMemBankSizeEightByte );
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_8), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_12), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_15), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_16), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_17), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_18), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size_18);
    hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_19), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_20), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size_64x128x8);
                            
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_9), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
        M = max_size, N = max_size, K = max_size;
        hipblasZgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_N, M, N, K, &alpha, (hipDoubleComplex*)dA, M, (hipDoubleComplex*)dB, K, &beta, (hipDoubleComplex*)dC_ref, M);
        
        if(kernel_number == 0)
        {   
            hipblasZgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_N, M, N, K, &alpha, (hipDoubleComplex*)dA, M, (hipDoubleComplex*)dB, K, &beta, (hipDoubleComplex*)dC, M);
        } 
        else if(kernel_number == 17){  
            dim3 blockDim(256);     
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 64));
            zgemm_17 <<<gridDim, blockDim, shared_mem_size>>>(M, N, K, dA, dB, dC, alpha, beta); 
        }    
        else if(kernel_number == 18){  
            dim3 blockDim(256);     
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 64));
            zgemm_18 <<<gridDim, blockDim, shared_mem_size_18>>>(M, N, K, dA, dB, dC, alpha, beta); 
        }   
        else if(kernel_number == 19){  
            dim3 blockDim(256);     
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 64));
            zgemm_19 <<<gridDim, blockDim, shared_mem_size>>>(M, N, K, dA, dB, dC, alpha, beta); 
        }   
        else if(kernel_number == 20){  
            dim3 blockDim(256);     
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 128));
            zgemm_20 <<<gridDim, blockDim, shared_mem_size_64x128x8>>>(M, N, K, dA, dB, dC, alpha, beta); 
        }   
  
         
        hipDeviceSynchronize();       
        hipMemcpy(C, dC, sizeof(double) * max_size * max_size * 2, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();  
        hipMemcpy(C_ref, dC_ref, sizeof(double) * max_size * max_size * 2, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();         
    
        // if (!verify_matrix_double2(C_ref, C, max_size)) {
        //     printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
        // exit(-3);            
        // }  
        hipEvent_t beg, end;   
        hipEventCreate(&beg); 
        hipEventCreate(&end);   
        float elapsed = 0;         
           
        if (kernel_number == 0){
            hipEventRecord(beg);
            for(int ii = 0; ii < num_tests; ++ii){
                    hipDeviceSynchronize();
                    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, (hipDoubleComplex*)dA, M, (hipDoubleComplex*)dB, K, &beta, (hipDoubleComplex*)dC, M);
                    hipDeviceSynchronize();
            }  
            hipEventRecord(end);
            hipEventSynchronize(beg);
            hipEventSynchronize(end);   
        } 
        else if(kernel_number == 17){
            hipEventRecord(beg); 
            dim3 blockDim(256); 
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 64));
            for(int ii = 0; ii < num_tests; ++ii){
                hipDeviceSynchronize();
                zgemm_17<<<gridDim, blockDim, shared_mem_size>>>(M, N, K, dA, dB, dC, alpha, beta);
                hipDeviceSynchronize();
            }
            hipEventRecord(end);
            hipEventSynchronize(beg); 
            hipEventSynchronize(end);
        } 
        else if(kernel_number == 18){
            hipEventRecord(beg);  
            dim3 blockDim(256); 
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 64));
            for(int ii = 0; ii < num_tests; ++ii){
                hipDeviceSynchronize();
                zgemm_18<<<gridDim, blockDim, shared_mem_size_18>>>(M, N, K, dA, dB, dC, alpha, beta);
                hipDeviceSynchronize();
            }
            hipEventRecord(end);
            hipEventSynchronize(beg); 
            hipEventSynchronize(end);
        } 
        else if(kernel_number == 19){
            hipEventRecord(beg); 
            dim3 blockDim(256); 
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 64));
            for(int ii = 0; ii < num_tests; ++ii){
                hipDeviceSynchronize();
                zgemm_19<<<gridDim, blockDim, shared_mem_size>>>(M, N, K, dA, dB, dC, alpha, beta);
                hipDeviceSynchronize();
            }
            hipEventRecord(end);
            hipEventSynchronize(beg); 
            hipEventSynchronize(end);
        } 
        else if(kernel_number == 20){
            hipEventRecord(beg); 
            dim3 blockDim(256); 
            dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 128));
            for(int ii = 0; ii < num_tests; ++ii){
                hipDeviceSynchronize();
                zgemm_20<<<gridDim, blockDim, shared_mem_size_64x128x8>>>(M, N, K, dA, dB, dC, alpha, beta);
                hipDeviceSynchronize();
            }    
            hipEventRecord(end);
            hipEventSynchronize(beg); 
            hipEventSynchronize(end);
        } 
 
        hipEventElapsedTime(&elapsed, beg, end);
        
        double gflops = double(8 * num_tests * double(M) * double(N) * double(K)) / (1e9);
        double perf = gflops / (elapsed / 1e3);
        printf("%8.2f,", perf);        
        fflush(stdout);
    }
    printf("\n");
}
