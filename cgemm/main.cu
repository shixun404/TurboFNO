#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.cuh"
#include "cgemm.cuh"

using DataT = float2;
int main(int argc, char** argv){
    DataT *A, *dA, *B, *dB, *C, *C_ref, *dC, *dC_ref;
    int M, N, K;
    M = atoi(argv[1]);
    N = atoi(argv[2]);
    K = atoi(argv[3]);
    // freopen("input.txt", "r", stdin);
    // scanf("%d%d%d", &M, &N, &K);
    long long int A_size = M * K;
    long long int B_size = N * K;
    long long int C_size = M * N;
    A = (DataT*)malloc(sizeof(DataT) * A_size);
    B = (DataT*)malloc(sizeof(DataT) * B_size);
    C = (DataT*)malloc(sizeof(DataT) * C_size);
    C_ref = (DataT*)malloc(sizeof(DataT) * C_size);

    hipMalloc((void**)&dA, sizeof(DataT) * A_size);
    hipMalloc((void**)&dB, sizeof(DataT) * B_size);
    hipMalloc((void**)&dC, sizeof(DataT) * C_size);
    hipMalloc((void**)&dC_ref, sizeof(DataT) * C_size);

    for(long long int i = 0; i < A_size; ++i) {
      A[i].x = float(rand() % 5) + (rand() % 5) * 0.01;
      A[i].y = float(rand() % 5) + (rand() % 5) * 0.01;
    }
    for(long long int i = 0; i < B_size; ++i){
      B[i].x = float(rand() % 5) + (rand() % 5) * 0.01;
      B[i].y = float(rand() % 5) + (rand() % 5) * 0.01;
    }
    for(long long int i = 0; i < C_size; ++i){
      C[i].x = float(rand() % 5) + (rand() % 5) * 0.01;
      C[i].y = float(rand() % 5) + (rand() % 5) * 0.01;
    }

    hipMemcpy(dA, A, sizeof(DataT) * A_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(DataT) * B_size, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice);
    hipMemcpy(dC_ref, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice);

    DataT alpha = {1.0, -1.0} , beta = {-1.0, 1.0}; 
    // DataT alpha = {1.0, 0} , beta = {1.0, 0}; 

    int num_tests = argc > 4 ? atoi(argv[4]) : 1;
   
    hipblasHandle_t handle;   
    hipblasCreate(&handle);
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, (hipFloatComplex*)&alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, (hipFloatComplex*)&beta, (hipFloatComplex*)dC_ref, M);     
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    float elapsed;
    hipDeviceSynchronize();
    hipEventRecord(beg);
    hipDeviceSynchronize();
    for(int i = 0; i < num_tests; ++i){
      hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, (hipFloatComplex*)&alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, (hipFloatComplex*)&beta, (hipFloatComplex*)dC_ref, M);
      hipDeviceSynchronize();
    }  
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed, beg, end);
    double gflops = (double(2 * num_tests * double(M) * double(N) * double(K)) / (1e9)) / (elapsed / 1e3);
    printf("cublas: %d, %d, %d, %f, %f\n", M, N, K, elapsed, gflops);


        
    {
      dim3 gridDim((M + THREADBLOCK_M - 1) / THREADBLOCK_M, (N + THREADBLOCK_N - 1) / THREADBLOCK_N, 1);
      dim3 blockDim(THREAD_NUM, 1, 1); 
      int shmem_size = sizeof(DataT) * (THREADBLOCK_M * THREADBLOCK_K + THREADBLOCK_N * THREADBLOCK_K) * 2;  
      printf("shmem_size=%d KB\n", shmem_size / 1024);
      size_t max_shmem;
// cudaFuncGetAttribute(&max_shmem, hipFuncAttributeMaxDynamicSharedMemorySize, cgemm);
// if (shmem_size > max_shmem) {
//     printf("Error: shmem_size=%d KB exceeds max available shared memory %lu KB\n", shmem_size / 1024, max_shmem / 1024);
// }
      cgemm<<<gridDim, blockDim, shmem_size>>>(M, N, K, dA, dB, dC, alpha, beta);
      hipError_t err = hipGetLastError();  // 获取最近的错误
if (err != hipSuccess) {
    printf("CUDA Kernel Launch Error: %s\n", hipGetErrorString(err));
}
hipDeviceSynchronize();  // 确保 kernel 执行完成
err = hipGetLastError();  // 再次检查同步后的错误
if (err != hipSuccess) {
    printf("CUDA Kernel Execution Error: %s\n", hipGetErrorString(err));
}

      hipEvent_t beg, end;
      hipEventCreate(&beg);
      hipEventCreate(&end);
      float elapsed;
      hipDeviceSynchronize();
      hipEventRecord(beg);
      hipDeviceSynchronize();
      for(int i = 0; i < num_tests; ++i){
          cgemm<<<gridDim, blockDim, shmem_size>>>(M, N, K, dA, dB, dC, alpha, beta);
          hipDeviceSynchronize();
      }  
      hipEventRecord(end);
      hipEventSynchronize(beg);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, beg, end);
      double gflops = (double(2 * num_tests * double(M) * double(N) * double(K)) / (1e9)) / (elapsed / 1e3);
      printf("cgemm: %d, %d, %d, %f, %f\n", M, N, K, elapsed, gflops);
  }

    hipMemcpy(C, dC, sizeof(DataT) * C_size, hipMemcpyDeviceToHost);
    hipMemcpy(C_ref, dC_ref, sizeof(DataT) * C_size, hipMemcpyDeviceToHost);

    verify_vector((float*)C_ref, (float*)C, M * N * 2);

    return 0;
}