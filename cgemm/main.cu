#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipblas.h>  
#include "utils/utils.cuh"
#include "cgemm.cuh"       
#define PPP 1    
#include <hip/hip_runtime.h> 
#include <helper_functions.h>           
#include <hip/hip_runtime_api.h>  
#define BM 64
#define BN 64
#define BK 8

int main(int argc, char **argv)
{        
    if (argc < 2) {  
        printf("Please select a kernel (range 0 - 1, here 0 is for NVIDIA cuBLAS).\n");
         exit(-1);
    }
    srand(10);  
    int kernel_number = atoi(argv[1]);
    int num_tests = 10;
    // const int NSPLIT = atoi(argv[4]);
    int start_size = atoi(argv[2]);   
    int end_size = atoi(argv[3]); 
    int gap_size = 256;
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
        printf("%8.2d|", max_size);
    }
    printf("\n");  
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
        printf("%8.2f|", min(8.1, float(max_size) * 31.25 / 1e3));
    } 

    printf("\n");
    // int threads_x = atoi(argv[4]); 
    float2 alpha, beta; 
    alpha.x = 1.0, alpha.y = 1.0;
	beta.x = 1.0, beta.y = 1.0;  
    int max_size = end_size;
    float *A = NULL, *B = NULL, *C_ref = NULL, *C = NULL;
    float *dA = NULL,*dB = NULL, *dC_ref = NULL, *dC = NULL;
    int size = max_size * sizeof (int);
    int deviceId;
    hipGetDevice(&deviceId); 
    hipDeviceProp_t props = getDetails(deviceId);
    
    A = (float *)malloc(sizeof(float) * max_size * max_size * 2);
    B = (float *)malloc(sizeof(float) * max_size * max_size * 2);
    C = (float *)malloc(sizeof(float) * max_size * max_size * 2);
    C_ref = (float *)malloc(sizeof(float) * max_size * max_size * 2);
    
    generate_random_matrix_float(A, max_size);
    generate_random_matrix_float(A + max_size * max_size, max_size);
     
    generate_random_matrix_float(B, max_size);
    generate_random_matrix_float(B + max_size * max_size, max_size);
    
    generate_random_matrix_float(C, max_size);
    generate_random_matrix_float(C + max_size * max_size, max_size);
    
    // for(int i = 0; i < max_size; ++i){
    //     for(int j = 0; j < max_size; ++j){  
    //         C[j + i * max_size] = (float)(j + i * max_size);
    //         C[j + i * max_size + max_size * max_size] = (float)(j + i * max_size);
    //         A[j * 2 + i * max_size * 2] = (float)j;
    //         A[j * 2 + i * max_size * 2 + 1] = (float)i;
    //         B[j * 2 + i * max_size * 2] = (float)j;
    //         B[j * 2 + i * max_size * 2 + 1] = (float)i;
 
    //     }
    // }
    copy_matrix_float(C, C_ref, max_size); 
    copy_matrix_float(C + max_size * max_size, C_ref + max_size * max_size , max_size);
   
    CUDA_CALLER(hipMalloc((void**) &dA, sizeof(float) * max_size * max_size * 2));
    CUDA_CALLER(hipMalloc((void**) &dB, sizeof(float) * max_size * max_size * 2));
    CUDA_CALLER(hipMalloc((void**) &dC, sizeof(float) * max_size * max_size * 2));
    CUDA_CALLER(hipMalloc((void**) &dC_ref, sizeof(float) * max_size * max_size * 2));
      
    CUDA_CALLER(hipMemcpy(dA, A, sizeof(float) * max_size * max_size * 2, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dB, B, sizeof(float) * max_size * max_size * 2, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC, C, sizeof(float) * max_size * max_size * 2, hipMemcpyHostToDevice));
    CUDA_CALLER(hipMemcpy(dC_ref, C_ref, sizeof(float) * max_size * max_size * 2, hipMemcpyHostToDevice));

    hipblasHandle_t handle;   
    hipblasCreate(&handle);      
      
    // if (!verify_matrix_float(C_ref, C, max_size) ||  
    //     !verify_matrix_float(C_ref + max_size * max_size, C + max_size * max_size, max_size)) {
    //     printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
    //     exit(-3);  
    // }    
    int M, N, K;        
    int shared_mem_size = ((BM + SKEW_KERNEL_2) * BK * 2 + (BN + SKEW_KERNEL_2) * BK * 2 ) * 2 * 8; 
    shared_mem_size = ((64 + 4) * 16 * 2) * 3 * 8;
    int shared_mem_size_18 = ((64 + 0) * 16 * 2) * 3 * 8;
    int shared_mem_size_64x128x8 = ((64 + 128) * 16) * 3 * 8;
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_20), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size_64x128x8);
                            
    // hipFuncSetAttribute(reinterpret_cast<const void*>(zgemm_9), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem_size);
    for(int max_size = start_size; max_size <= end_size; max_size += gap_size){
        M = max_size, N = max_size, K = max_size;
        hipblasCgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_N, M, N, K, &alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, &beta, (hipFloatComplex*)dC_ref, M);
        
        if(kernel_number == 0)
        {   
            hipblasCgemm(handle, HIPBLAS_OP_N,HIPBLAS_OP_N, M, N, K, &alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, &beta, (hipFloatComplex*)dC, M);
        } 
        // else if(kernel_number == 20){  
        //     dim3 blockDim(256);     
        //     dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 128));
        //     zgemm_20 <<<gridDim, blockDim, shared_mem_size_64x128x8>>>(M, N, K, dA, dB, dC, alpha, beta); 
        // }   
  
         
        hipDeviceSynchronize();       
        hipMemcpy(C, dC, sizeof(float) * max_size * max_size * 2, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();  
        hipMemcpy(C_ref, dC_ref, sizeof(float) * max_size * max_size * 2, hipMemcpyDeviceToHost);
        hipDeviceSynchronize();         
    
        // if (!verify_matrix_float2(C_ref, C, max_size)) {
        //     printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
        // exit(-3);            
        // }  
        hipEvent_t beg, end;   
        hipEventCreate(&beg); 
        hipEventCreate(&end);   
        float elapsed = 0;         
           
        if (kernel_number == 0){
            hipEventRecord(beg);
            for(int ii = 0; ii < num_tests; ++ii){
                    hipDeviceSynchronize();
                    hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, &beta, (hipFloatComplex*)dC, M);
                    hipDeviceSynchronize();
            }  
            hipEventRecord(end);
            hipEventSynchronize(beg);
            hipEventSynchronize(end);   
        } 
        // else if(kernel_number == 20){
        //     hipEventRecord(beg); 
        //     dim3 blockDim(256); 
        //     dim3 gridDim(CEIL_DIV(max_size, 64), CEIL_DIV(max_size, 128));
        //     for(int ii = 0; ii < num_tests; ++ii){
        //         hipDeviceSynchronize();
        //         zgemm_20<<<gridDim, blockDim, shared_mem_size_64x128x8>>>(M, N, K, dA, dB, dC, alpha, beta);
        //         hipDeviceSynchronize();
        //     }    
        //     hipEventRecord(end);
        //     hipEventSynchronize(beg); 
        //     hipEventSynchronize(end);
        // } 
 
        hipEventElapsedTime(&elapsed, beg, end);
        
        float gflops = float(8 * num_tests * float(M) * float(N) * float(K)) / (1e9);
        float perf = gflops / (elapsed / 1e3);
        printf("%8.2f,", perf);        
        fflush(stdout);
    }
    printf("\n");
}
