#include "hip/hip_runtime.h"

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.cuh"
#include "cgemm.cuh"

using DataT = float2;
int main(int argc, char** argv){
    DataT *A, *dA, *B, *dB, *C, *C_ref, *dC, *dC_ref;
    int M, N, K;
    freopen("input.txt", "r", stdin);
    scanf("%d%d%d", &M, &N, &K);
    long long int A_size = ((M + 127) / 128) * 128 * ((K + 127) / 128) * 128;
    long long int B_size = ((N + 127) / 128) * 128 * ((K + 127) / 128) * 128;
    long long int C_size = ((M + 127) / 128) * 128 * ((N + 127) / 128) * 128;
    A = (DataT*)malloc(sizeof(DataT) * A_size);
    B = (DataT*)malloc(sizeof(DataT) * B_size);
    C = (DataT*)malloc(sizeof(DataT) * C_size);
    C_ref = (DataT*)malloc(sizeof(DataT) * C_size);

    hipMalloc((void**)&dA, sizeof(DataT) * A_size);
    hipMalloc((void**)&dB, sizeof(DataT) * B_size);
    hipMalloc((void**)&dC, sizeof(DataT) * C_size);
    hipMalloc((void**)&dC_ref, sizeof(DataT) * C_size);

    for(long long int i = 0; i < A_size; ++i) {
      A[i].x = float(rand() % 5) + (rand() % 5) * 0.01;
      A[i].y = float(rand() % 5) + (rand() % 5) * 0.01;
    }
    for(long long int i = 0; i < B_size; ++i){
      B[i].x = float(rand() % 5) + (rand() % 5) * 0.01;
      B[i].y = float(rand() % 5) + (rand() % 5) * 0.01;
    }
    for(long long int i = 0; i < C_size; ++i){
      C[i].x = float(rand() % 5) + (rand() % 5) * 0.01;
      C[i].y = float(rand() % 5) + (rand() % 5) * 0.01;
    }

    hipMemcpy(dA, A, sizeof(DataT) * A_size, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(DataT) * B_size, hipMemcpyHostToDevice);
    hipMemcpy(dC, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice);
    hipMemcpy(dC_ref, C, sizeof(DataT) * C_size, hipMemcpyHostToDevice);

    DataT alpha = {0.1,0.1} , beta = {0.1,0.1}; 

    int num_tests = 1;

    cgemm(M, N, K, dA, dB, dC, alpha, beta);
    
    hipblasHandle_t handle;   
    hipblasCreate(&handle);
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, (hipFloatComplex*)&alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, (hipFloatComplex*)&beta, (hipFloatComplex*)dC, M);     
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);
    float elapsed;
    hipDeviceSynchronize();
    hipEventRecord(beg);
    hipDeviceSynchronize();
    for(int i = 0; i < 10; ++i){
      hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, (hipFloatComplex*)&alpha, (hipFloatComplex*)dA, M, (hipFloatComplex*)dB, K, (hipFloatComplex*)&beta, (hipFloatComplex*)dC_ref, M);
      hipDeviceSynchronize();
    }  
    hipEventRecord(end);
    hipEventSynchronize(beg);
    hipEventSynchronize(end);
    hipEventElapsedTime(&elapsed, beg, end);
    double gflops = (double(2 * 10 * double(M) * double(N) * double(K)) / (1e9)) / (elapsed / 1e3);
    printf("%d, %d, %d, %f, %f\n", M, N, K, elapsed, gflops);


    hipMemcpy(C, dC, sizeof(DataT) * C_size, hipMemcpyDeviceToHost);
    hipMemcpy(C_ref, dC_ref, sizeof(DataT) * C_size, hipMemcpyDeviceToHost);

    verify_vector((float*)C_ref, (float*)C, M * N * 2);

    return 0;
}